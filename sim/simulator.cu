#include "hip/hip_runtime.h"
#include <stdint.h>
#include <string.h>
#include <stdlib.h> 
#include <iostream>
#include <sstream>
#include <fstream>
#include <vector>
#include "Tokenizer.h"
#include "gateMatrix.h" 
using namespace std;

// GLobal functions 
void SimulateOnCuda(gateMatrix* matrix, LogicValue* input, LogicValue* ouput, uint32_t num_passes);
__global__ void Simulate(uint64_t* matrix, uint32_t num_row, uint32_t num_col, 
                         LogicValue* input, uint32_t num_inp,   
                         LogicValue* output, uint32_t num_out, uint32_t num_passes);
gateMatrix* createMatrixForCuda(void); 
void getInput(char* inputFile, gateMatrix* matrix, LogicValue* input, uint32_t* num_passes);
void printOutput(char* outputFile, gateMatrix* matrix, LogicValue* output, uint32_t num_passes);

// Files for input and output
char* inputFile = "input.txt";
char* outputFile = "output.txt";


int main(void){ 
  LogicValue *input, *output; // given inputs and produced ouputs
  uint32_t *num_passes;       // number of inputs passes to iterate over
 
  // reserving space to create matrix from graph, <Design.h>
  gateMatrix* matrix = createMatrixForCuda();
 
  // parse input file
  getInput(inputFile, matrix, input, num_passes);

  // simulate design
  SimulateOnCuda(matrix, input, output, *num_passes);

  // print output to file
  printOutput(outputFile, matrix, output, *num_passes);

  // deallocate memory
  delete input;
  delete output;
  delete matrix;
}



// Initialize Memory to simulate for Cuda
void SimulateOnCuda(gateMatrix* matrix, LogicValue* input, LogicValue* output, uint32_t num_passes){
  // Initialize pointers for cuda memory
  uint64_t *d_matrix;
  LogicValue *d_input, *d_output;
  uint32_t mat_size = matrix->getNumRow() * matrix->getNumCol() * sizeof(uint64_t);
  uint32_t inp_size = matrix->getNumInp() * num_passes * sizeof(LogicValue);
  uint32_t out_size = matrix->getNumOut() * num_passes * sizeof(LogicValue);

  // Allocate space for device copies
  hipMalloc((void**)&d_matrix, mat_size);
  hipMalloc((void**)&d_input, inp_size);
  hipMalloc((void**)&d_output, out_size);

  // Copy matrix and inputs to device
  hipMemcpy(d_matrix, matrix->getRawMatrix(), mat_size, hipMemcpyHostToDevice);
  hipMemcpy(d_input, input, inp_size, hipMemcpyHostToDevice); 

#ifdef PRINTALL
  //int iterations = num_passes;
  //num_passes = 1;
  //for(int i = 0; i < iterations; i++){
#endif    
  // Launch Kernel on GPU
  Simulate<<<1, matrix->getNumCol(), mat_size>>>(d_matrix, matrix->getNumRow(), matrix->getNumCol(),
                                                 d_input, inp_size, d_output, out_size, num_passes);

  // Copy matrix and results back to host
  hipMemcpy(matrix->getRawMatrix(), d_matrix, mat_size, hipMemcpyDeviceToHost); 
  hipMemcpy(output, d_output, out_size, hipMemcpyDeviceToHost);

#ifdef PRINTALL
  //}
#endif
} 

 
// when simulating with multiple inputs, try to not leave here..
// so setup a shared memory gate representation and work here... .
__global__ void Simulate(uint64_t* matrix, uint32_t num_row, uint32_t num_col, 
                         LogicValue* input, uint32_t num_inp,   
                         LogicValue* output, uint32_t num_out, uint32_t num_passes){ 
  extern __shared__ uint64_t sMatrix[];
  // int myId = threadIdx.x +blockDim.x * blockIdx.x;
  uint32_t tid = threadIdx.x; // TODO num_col == block? 
  uint64_t gateEntry;
  int gateInp0, gateInp1, gateOut;

  // move gate network into shared memory
  for(uint32_t i = 0; i < num_row; i++){    
    sMatrix[i * num_col +  tid] =  matrix[i * num_col + tid];
    __syncthreads();
  }

  // enter input values (0) 
  if(tid < num_inp){
    sMatrix[tid] &= (~OUT_MASK);
    sMatrix[tid] |= setOUT(input[tid]); // TODO will need to fix based on location of input..
    __syncthreads();
  } 

  // evaluate circuit (0 -> num_row - 1)
  for(uint32_t i = 1; i < num_row; i++){
    gateEntry = sMatrix[i * num_col + tid];    
    gateInp0  = (LogicValue)getOUT(sMatrix[getI0R(gateEntry) * num_col + getI0C(gateEntry)]); 
    gateInp1  = (LogicValue)getOUT(sMatrix[getI1R(gateEntry) * num_col + getI1C(gateEntry)]);

    // TODO find a way to simplify?
    switch(getGATE(gateEntry)){
      case NO_GATE:
        gateOut = 0;
        break;
      case PORT_I:
        break;
      case PORT_O:
      case OBUF:
        gateOut = gateInp0; 
        break;
      case RTL_INV: // TODO for all gates
        switch(gateInp0){
          case O:
            gateOut = I;
            break;
          case I: 
            gateOut = O;
            break;
          case X:
            gateOut = X;
            break;
          case Z:
            gateOut = Z;
            break;
        }
        break;
      case RTL_AND:
        gateOut = gateInp0 & gateInp1;
        break;
      case RTL_OR:
        gateOut = gateInp0 | gateInp1;
        break;
      case RTL_XOR: // only works for 0 and 1
        gateOut = gateInp0 ^ gateInp1;
        break;
      case RTL_NAND:
        gateOut = !(gateInp0 & gateInp1);
        break;
      case RTL_NOR:
        gateOut = !(gateInp0 | gateInp1);
        break;
      default:
        break;
    }
    sMatrix[i * num_col + tid] &= (~OUT_MASK);
    sMatrix[i * num_col + tid] |= setOUT(gateOut);
    __syncthreads(); 
  } 

  // test code
   for(uint32_t i = 0; i < num_row; i++){    
    matrix[i * num_col +  tid] =  sMatrix[i * num_col + tid];
    __syncthreads();
  } 

  // enter output values 
  if(tid < num_out){
    output[tid] = (LogicValue)setOUT(sMatrix[(num_row - 1) * num_col + tid]);
  }
}


/* HELPER FUNCTIONS */

// create Matrix for Cuda from Design header file (DESIGN.h) 
gateMatrix* createMatrixForCuda(void){

  gateMatrix* matrix = new gateMatrix(CUDA_MATRIX_ROW, CUDA_MATRIX_COL, 
                                      CUDA_MATRIX_INP, CUDA_MATRIX_OUT);
  for(int i = 0; i < CUDA_MATRIX_ROW; i++){
    for(int j = 0; j < CUDA_MATRIX_COL; j++){
      matrix->addGate(CUDA_MATRIX[i][j], i, j);
    }
  } 
  return matrix;
}

void getInput(char* inputFile, gateMatrix* matrix, LogicValue* input, uint32_t* num_passes){
  ifstream file;
  file.open(inputFile);
  if(!file){
    cout << "Error: Can't open the file.\n";
    exit(1); 
  }
  
  string str; const char* delim = " ";

  // get number of passes
  if(getline(file,str)){
    char* str_c = strdup(str.c_str());
    char* token = strtok(str_c, delim);
    *num_passes = atoi(token);
cout << "" << *num_passes;
    free(str_c);
  }
  
  input = new LogicValue[matrix->getNumInp() *  (*num_passes)];

  // get inputs
  for(int i = 0; i < *num_passes; i++){
    int j = 0;
    while(getline(file,str)){
      char* str_c = strdup(str.c_str());
      char* token = strtok(str_c, delim);
      input[i * matrix->getNumInp() + j] = (LogicValue)atoi(token);
cout << "" << input[i * matrix->getNumInp() + j];
     j++;
    }
  }

  file.close();
  exit(1);
}

void printOutput(char* outputFile, gateMatrix* matrix, LogicValue* output, uint32_t num_passes){

}


/*  Tokenizer T = Tokenizer(' ', ' ');

  int getI(string &input){
    int out;
    stringstream myStream(input);
    if(!(myStream >> out))
      cout << "getI broken";
    return out;
  } 

  LogicValue* getUserInput(gateMatrix* matrix){
    string in; string** token;
    LogicValue* input = new LogicValue[matrix->getNumInp()];  
 
    cout << "Enter input of size " << matrix->getNumInp() << ". (ex: 0 1 0 0 1)";
    getline(cin, in); 
    token = T.tokenize(&in);
    for(int i = 0; i < matrix->getNumInp(); i++){
      input[i] = (LogicValue)getI(*token[i]); 
      cout << "" << input[i];
    }
    return input; 
  }
  //printUserOutput();
*/
